#include "hip/hip_runtime.h"
#include <stdio.h>
#include "RBased_DWT_common.h"



/*


Functions in this code are organized in 7 sections. 2 Sections corresponds to functions executed in the host and the other 5 are functions used in the device. 

Host functions sections:

1. PRE/POST COMPUTE FUNCTIONS: Allocate and release device memory.
2. DWT FUNCTIONS: Precomputing needed to apply N levels of the DWT over an input, launch a kernel for each DWT level.

Device functions sections:

3. CUDA KERNELS: Each CUDA kernel computes a single DWT level (vertical + horizontal filter).
4. PRE-COMPUTE FUNCTIONS: Inline device functions used by the kernels to (mainly) compute in which input coordinates each warp has to work (asign a data block to a warp).
5. DATA MANAGEMENT FUNCTIONS: Inline device functions used by the kernels to read (or write) a data block from the device main memory to the registers, and the other way around.
6. FILTER COMPUTATION FUNCTIONS: Inline device functions used by the kernels to compute the vertical or horizontal filter over a full data block.
7. FILTER KERNEL FUNCTIONS:  Inline device functions used by the kernels to compute a lifting step operation over 3 samples.		


Example of function call flow:

	<HOST> PRE/POST COMPUTE FUNCTIONS 
	|
	|__ <HOST> DWT FUNCTIONS 
		|
		|__ <DEVICE> CUDA KERNELS 	
			|
			|__ <DEVICE> PRE-COMPUTE FUNCTIONS 	
				<DEVICE> DATA MANAGEMENT FUNCTIONS 		
				<DEVICE> FILTER COMPUTATION FUNCTIONS 	
				|
				|__ <DEVICE> FILTER KERNEL FUNCTIONS 				

*/




/**************************************************************

START - <DEVICE> FILTER KERNEL FUNCTIONS

**************************************************************/

//CDF 5/3 (1st Lifting Step) - FORWARD 
inline __device__ void LStep_1_53_F(int a, VOLATILE int* b, int c){ *b -= ((a+c)>>1);}

//CDF 5/3 (2nd Lifting Step) - FORWARD 
inline __device__ void LStep_2_53_F(int a, VOLATILE int* b, int c){ *b += ((a + c + 2)>>2);}
		

//CDF 5/3 (1st Lifting Step) - REVERSE
inline __device__ void LStep_1_53_R(int a, VOLATILE int* b, int c){ *b -= (((a + c + 2))>>2);}

//CDF 5/3 (2nd Lifting Step) - REVERSE
inline __device__ void LStep_2_53_R(int a, VOLATILE int* b, int c){ *b += (((a+c)>>1));}



//CDF 9/7 (1st Lifting Step) - FORWARD 
inline __device__ void LStep_1_97_F(float a, VOLATILE float* b, float c){ *b += ((a + c)* LIFTING_STEPS_I97_1);}

//CDF 9/7 (2nd Lifting Step) - FORWARD 
inline __device__ void LStep_2_97_F(float a, VOLATILE float* b, float c){ *b += ((a + c)* LIFTING_STEPS_I97_2);}

//CDF 9/7 (3rd Lifting Step) - FORWARD 
inline __device__ void LStep_3_97_F(float a, VOLATILE float* b, float c){ *b += ((a + c)* LIFTING_STEPS_I97_3);}

//CDF 9/7 (4th Lifting Step) - FORWARD + normalization
inline __device__ void LStep_4_97_F(float a, VOLATILE float* b, float c){ *b = (*b + ((a + c)* LIFTING_STEPS_I97_4))*NORMALIZATION_I97_2;}	
		


//CDF 9/7 (1st Lifting Step) - REVERSE + normalization
inline __device__ void LStep_1_97_R(float a, VOLATILE float* b, float c){ *b = (*b/NORMALIZATION_I97_2) - ((a + c)* LIFTING_STEPS_I97_4);}

//CDF 9/7 (2nd Lifting Step) - REVERSE
inline __device__ void LStep_2_97_R(float a, VOLATILE float* b, float c){ *b -= ((a + c)* LIFTING_STEPS_I97_3);}

//CDF 9/7 (3rd Lifting Step) - REVERSE
inline __device__ void LStep_3_97_R(float a, VOLATILE float* b, float c){ *b -= ((a + c)* LIFTING_STEPS_I97_2);}

//CDF 9/7 (4th Lifting Step) - REVERSE
inline __device__ void LStep_4_97_R(float a, VOLATILE float* b, float c){ *b -= ((a + c)* LIFTING_STEPS_I97_1);}

//END - <DEVICE> FILTER KERNEL FUNCTIONS -----------------------------------------------------------------------













/**************************************************************
//START - <DEVICE> FILTER COMPUTATION FUNCTIONS
**************************************************************/



//VERTICAL FILTER FUNCTIONS , generic for all versions <shuffle instructions, shared memory with auxiliary buffer or full shared memory>	




inline __device__ void Vertical_Filter_Forward_53(int* TData,int TDSize_Y, int TDSize_X)
{
	int TDSize_Y_index	= 0;

	for(int TDSize_X_index = 0; TDSize_X_index < TDSize_X; TDSize_X_index++)
	{	
		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{			
			LStep_1_53_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}			
		
		LStep_1_53_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)]);		
		TDSize_Y_index = 0;
		LStep_2_53_F(TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)]);
		
		for(TDSize_Y_index = 2; TDSize_Y_index < TDSize_Y; TDSize_Y_index += 2)
		{
			LStep_2_53_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)]);
		}			
	}
}

inline __device__ void Vertical_Filter_Reverse_53(int* TData,int TDSize_Y, int TDSize_X)
{
	int TDSize_Y_index	= 0;
	
	for(int TDSize_X_index = 0; TDSize_X_index < TDSize_X; TDSize_X_index++)
	{			
		TDSize_Y_index	= 0;
		LStep_1_53_R(TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		
		for(TDSize_Y_index = 2; TDSize_Y_index < TDSize_Y; TDSize_Y_index += 2)
		{
			LStep_1_53_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}

		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{			
			LStep_2_53_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}				
		
		LStep_2_53_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);			
	}
}

inline __device__ void Vertical_Filter_Forward_97(float* TData, int TDSize_Y, int TDSize_X)
{

	int TDSize_Y_index	= 0;

	for(int TDSize_X_index = 0; TDSize_X_index < TDSize_X; TDSize_X_index++)
	{	
		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{
			LStep_1_97_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}
						
		LStep_1_97_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		TDSize_Y_index = 0;
		LStep_2_97_F(TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		
		for(TDSize_Y_index = 2; TDSize_Y_index < TDSize_Y; TDSize_Y_index += 2)
		{
			LStep_2_97_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}

		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{
			LStep_3_97_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}			
		
		LStep_3_97_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);		
		TDSize_Y_index = 0;
		LStep_4_97_F(TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);

		
		for(TDSize_Y_index = 2; TDSize_Y_index < TDSize_Y; TDSize_Y_index += 2)
		{
			LStep_4_97_F(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}

		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{
			TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))] *= NORMALIZATION_I97_1;
		}

		TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))] *= NORMALIZATION_I97_1;			
	}
}

inline __device__ void Vertical_Filter_Reverse_97(float* TData, int TDSize_Y, int TDSize_X)
{

	int TDSize_Y_index	= 0;

	for(int TDSize_X_index = 0; TDSize_X_index < TDSize_X; TDSize_X_index++)
	{
		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{
			TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))] /= NORMALIZATION_I97_1;
		}

		TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))] /= NORMALIZATION_I97_1;
		TDSize_Y_index	= 0;
		LStep_1_97_R(TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		
		for(TDSize_Y_index = 2; TDSize_Y_index < TDSize_Y; TDSize_Y_index += 2)
		{
			LStep_1_97_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}

		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{			
			LStep_2_97_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}				
		
		LStep_2_97_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);		
		TDSize_Y_index	= 0;
		LStep_3_97_R(TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		
		for(TDSize_Y_index = 2; TDSize_Y_index < TDSize_Y; TDSize_Y_index += 2)
		{
			LStep_3_97_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}	
		
		for(TDSize_Y_index = 1; TDSize_Y_index < (TDSize_Y-1); TDSize_Y_index += 2)
		{			
			LStep_4_97_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index+1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);
		}				
		
		LStep_4_97_R(TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))], TData[((TDSize_Y_index-1)*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*(threadIdx.x))]);					
	}
}




//HORIZONTAL FILTER FUNCTIONS , specific for each version <shuffle instructions, shared memory with auxiliary buffer or full shared memory>		


//SHARED MEMORY W/ AUXILIARY BUFFER - HORIZONTAL FILTER FUNCTIONS


inline __device__ void Horizontal_Filter_Forward_53_Shared(int* TData, int* Shared_Data, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){	
		Shared_Data[threadIdx.x] = TData[TDSize_Y_index*2];
		LStep_1_53_F(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], Shared_Data[(((threadIdx.x)%32)==31?threadIdx.x:threadIdx.x+1)]);
	}

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){		
		Shared_Data[threadIdx.x] = TData[(TDSize_Y_index*2)+1];
		LStep_2_53_F(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], Shared_Data[(((threadIdx.x)%32)==0?threadIdx.x:threadIdx.x-1)]);
	}
}

inline __device__ void Horizontal_Filter_Reverse_53_Shared(int* TData, int* Shared_Data, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){	
		Shared_Data[threadIdx.x] = TData[(TDSize_Y_index*2)+1];
		LStep_1_53_R(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], Shared_Data[(((threadIdx.x)%32)==0?threadIdx.x:threadIdx.x-1)]);
	}

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){
		Shared_Data[threadIdx.x] = TData[TDSize_Y_index*2];
		LStep_2_53_R(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], Shared_Data[(((threadIdx.x)%32)==31?threadIdx.x:threadIdx.x+1)]);
	}
}

inline __device__ void Horizontal_Filter_Forward_97_Shared(float* TData, float* Shared_Data, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){

		Shared_Data[threadIdx.x] = TData[TDSize_Y_index*2];
		LStep_1_97_F(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], Shared_Data[(((threadIdx.x)%32)==31?threadIdx.x:threadIdx.x+1)]);
	
		Shared_Data[threadIdx.x] = TData[(TDSize_Y_index*2)+1];
		LStep_2_97_F(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], Shared_Data[(((threadIdx.x)%32)==0?threadIdx.x:threadIdx.x-1)]);

		Shared_Data[threadIdx.x] = TData[TDSize_Y_index*2];	
		LStep_3_97_F(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], Shared_Data[(((threadIdx.x)%32)==31?threadIdx.x:threadIdx.x+1)]);

		Shared_Data[threadIdx.x] = TData[(TDSize_Y_index*2)+1];
		LStep_4_97_F(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], Shared_Data[(((threadIdx.x)%32)==0?threadIdx.x:threadIdx.x-1)]);

		TData[(TDSize_Y_index*2)+TDSize_X_index] *= NORMALIZATION_I97_1;
	}		
}

inline __device__ void Horizontal_Filter_Reverse_97_Shared(float* TData, float* Shared_Data, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){
	
		TData[(TDSize_Y_index*2)+TDSize_X_index] /= NORMALIZATION_I97_1;

		Shared_Data[threadIdx.x] = TData[(TDSize_Y_index*2)+1];	
		LStep_1_97_R(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], Shared_Data[(((threadIdx.x)%32)==0?threadIdx.x:threadIdx.x-1)]);

		Shared_Data[threadIdx.x] = TData[TDSize_Y_index*2];
		LStep_2_97_R(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], Shared_Data[(((threadIdx.x)%32)==31?threadIdx.x:threadIdx.x+1)]);

		Shared_Data[threadIdx.x] = TData[(TDSize_Y_index*2)+1];
		LStep_3_97_R(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], Shared_Data[(((threadIdx.x)%32)==0?threadIdx.x:threadIdx.x-1)]);

		Shared_Data[threadIdx.x] = TData[TDSize_Y_index*2];
		LStep_4_97_R(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], Shared_Data[(((threadIdx.x)%32)==31?threadIdx.x:threadIdx.x+1)]);	
	}	
}



//FULL SHARED MEMORY - HORIZONTAL FILTER FUNCTIONS




inline __device__ void Horizontal_Filter_Forward_53_Full_Shared(VOLATILE int* TData, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)
		LStep_1_53_F(TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], &(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)]), TData[(TDSize_Y_index*2)+(((threadIdx.x)%32)==31?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x+1)))]);

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)	
		LStep_2_53_F(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+TDSize_X_index+(((threadIdx.x)%32)==0?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x-1)))]);

}

inline __device__ void Horizontal_Filter_Reverse_53_Full_Shared(VOLATILE int* TData, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)	
		LStep_1_53_R(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+TDSize_X_index+(((threadIdx.x)%32)==0?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x-1)))]);

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)
		LStep_2_53_R(TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+(((threadIdx.x)%32)==31?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x+1)))]);
}

inline __device__ void Horizontal_Filter_Forward_97_Full_Shared(VOLATILE float* TData, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){

		LStep_1_97_F(TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], &(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)]), TData[(TDSize_Y_index*2)+(((threadIdx.x)%32)==31?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x+1)))]);
		LStep_2_97_F(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+TDSize_X_index+(((threadIdx.x)%32)==0?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x-1)))]);
		LStep_3_97_F(TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], &(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)]), TData[(TDSize_Y_index*2)+(((threadIdx.x)%32)==31?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x+1)))]);
		LStep_4_97_F(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+TDSize_X_index+(((threadIdx.x)%32)==0?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x-1)))]);
	
		TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)] *= NORMALIZATION_I97_1;
	}	
}

inline __device__ void Horizontal_Filter_Reverse_97_Full_Shared(VOLATILE float* TData, int TDSize_Y, int TDSize_X)
{
	int TDSize_X_index = TDSize_X>>1;

	for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){
	
		TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)] /= NORMALIZATION_I97_1;

		LStep_1_97_R(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+TDSize_X_index+(((threadIdx.x)%32)==0?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x-1)))]);
		LStep_2_97_R(TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+(((threadIdx.x)%32)==31?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x+1)))]);
		LStep_3_97_R(TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+TDSize_X_index+(((threadIdx.x)%32)==0?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x-1)))]);
		LStep_4_97_R(TData[(TDSize_Y_index*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], &TData[(TDSize_Y_index*2)+TDSize_X_index+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[(TDSize_Y_index*2)+(((threadIdx.x)%32)==31?(SHARED_MEMORY_STRIDE*threadIdx.x):(SHARED_MEMORY_STRIDE*(threadIdx.x+1)))]);	
	}		
}




//SHUFFLE INSTRUCTIONS - HORIZONTAL FILTER FUNCTIONS




#if SHUFFLE == 1	

	inline __device__ void Horizontal_Filter_Forward_53_Shuffle(int* TData, int TDSize_Y, int TDSize_X)
	{
		int TDSize_X_index = TDSize_X>>1;

		for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)
			LStep_1_53_F(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], __shfl_down(TData[TDSize_Y_index*2], 1));

		for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)
			LStep_2_53_F(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], __shfl_up(TData[(TDSize_Y_index*2)+TDSize_X_index], 1));
	}

	inline __device__ void Horizontal_Filter_Reverse_53_Shuffle(int* TData, int TDSize_Y, int TDSize_X)
	{
		int TDSize_X_index = TDSize_X>>1;

		for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)		
			LStep_1_53_R(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], __shfl_up(TData[(TDSize_Y_index*2)+TDSize_X_index], 1));

		for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++)
			LStep_2_53_R(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], __shfl_down(TData[TDSize_Y_index*2], 1));	
	}

	inline __device__ void Horizontal_Filter_Forward_97_Shuffle(float* TData, int TDSize_Y, int TDSize_X)
	{
		int TDSize_X_index = TDSize_X>>1;

		for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){
			LStep_1_97_F(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], __shfl_down(TData[TDSize_Y_index*2], 1));
			LStep_2_97_F(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], __shfl_up(TData[(TDSize_Y_index*2)+TDSize_X_index], 1));
			LStep_3_97_F(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], __shfl_down(TData[TDSize_Y_index*2], 1));
			LStep_4_97_F(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], __shfl_up(TData[(TDSize_Y_index*2)+TDSize_X_index], 1));
	
			TData[(TDSize_Y_index*2)+TDSize_X_index] *= NORMALIZATION_I97_1;
		}		
	}

	inline __device__ void Horizontal_Filter_Reverse_97_Shuffle(float* TData, int TDSize_Y, int TDSize_X)
	{
		int TDSize_X_index = TDSize_X>>1;

		for(int TDSize_Y_index = 0; TDSize_Y_index < TDSize_Y; TDSize_Y_index ++){
		
			TData[(TDSize_Y_index*2)+TDSize_X_index] /= NORMALIZATION_I97_1;

			LStep_1_97_R(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], __shfl_up(TData[(TDSize_Y_index*2)+TDSize_X_index], 1));
			LStep_2_97_R(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], __shfl_down(TData[TDSize_Y_index*2], 1));
			LStep_3_97_R(TData[(TDSize_Y_index*2)+TDSize_X_index], &TData[TDSize_Y_index*2], __shfl_up(TData[(TDSize_Y_index*2)+TDSize_X_index], 1));
			LStep_4_97_R(TData[TDSize_Y_index*2], &TData[(TDSize_Y_index*2)+TDSize_X_index], __shfl_down(TData[TDSize_Y_index*2], 1));	
		}		
	}

#endif

//END - <DEVICE> FILTER COMPUTATION FUNCTIONS -----------------------------------------------------------------------














//START - <DEVICE> DATA MANAGEMENT FUNCTIONS -----------------------------------------------------------------------

inline __device__ void UpdateSubbandsCoordinates(int DSize_Current_X, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH)
{
	*TCoordinate_LL += DSize_Current_X;
	*TCoordinate_HL += DSize_Current_X;
	*TCoordinate_LH += DSize_Current_X;
}

inline __device__ void UpdateSubbandsCoordinates_LLaux(int DSize_Current_X, int DSize_Initial_X, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH)
{
	*TCoordinate_LL += (DSize_Current_X>>1);
	*TCoordinate_HL += DSize_Initial_X;
	*TCoordinate_LH += DSize_Initial_X;
}

inline __device__ void UpdateSubbandsCoordinates_Scheduler(int DSize_Current_X, int DSize_Initial_X, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int Last_Level)
{
	if(Last_Level)	UpdateSubbandsCoordinates(DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH);
	else			UpdateSubbandsCoordinates_LLaux(DSize_Current_X, DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH);
}

inline __device__ void ReadBlock(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate)
{

	for(int y = 0; y < TDSize_Y; y++){

		TData[(y<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data[*TCoordinate];
		TData[(y<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data[(*TCoordinate)+1];

		*TCoordinate += DSize_Current_X;
	}
}

inline __device__ void ReadBlock2(DATATYPE2* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate)
{

	*(TCoordinate)>>=1;
	for(int y = 0; y < TDSize_Y; y++){
		
		#if DATATYPE_16BITS_or_32BITS == 1

			TData[(y*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = 		Data[*TCoordinate];
			TData[(y*2)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = 	((int)TData[(y*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)])>>16;	
			TData[(y*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = 		(DATATYPE)TData[(y*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)];		
		
		#else

			TData[(y*2)] = 		Data[*TCoordinate].x;
			TData[(y*2)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = 	Data[*TCoordinate].y;
			
		#endif

		*TCoordinate += DSize_Current_X;
	}
	*(TCoordinate)<<=1;
}

inline __device__ void WriteSubbands(DATATYPE* Data, int DSize_Initial_X, int DSize_Current_X, REG_DATATYPE* TData, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int* index, int Last_Level){

	Data[*TCoordinate_LL] = TData[((*index)<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)];
	Data[*TCoordinate_HL] = TData[((*index)<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)];

	++(*index);

	Data[*TCoordinate_LH] = TData[((*index)<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)];
	Data[*TCoordinate_LH + (DSize_Current_X>>1)] = TData[((*index)<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)];

	UpdateSubbandsCoordinates_Scheduler(DSize_Current_X, DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, Last_Level);
}

inline __device__ void WriteSubbands_Top(DATATYPE* Data, int DSize_Current_X,  int DSize_Initial_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int Last_Level, int Overlap)
{	
	for(int y = 0; y < (TDSize_Y - (Overlap>>1)); y++)	
		WriteSubbands(Data, DSize_Initial_X, DSize_Current_X, TData, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, &y, Last_Level);			
}

inline __device__ void WriteSubbands_Middle(DATATYPE* Data, int DSize_Current_X,  int DSize_Initial_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int Last_Level, int Overlap)
{	
	for(int y = 0; y < (Overlap>>1); y+=2)
		UpdateSubbandsCoordinates_Scheduler(DSize_Current_X, DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, Last_Level);
	
	for(int y = (Overlap>>1); y < (TDSize_Y - (Overlap>>1)); y++)
		WriteSubbands(Data, DSize_Initial_X, DSize_Current_X, TData, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, &y, Last_Level);	
}

inline __device__ void WriteSubbands_Bottom(DATATYPE* Data, int DSize_Current_X,  int DSize_Initial_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int Last_Level, int Overlap)
{	
	for(int y = 0; y < (Overlap>>1); y+=2)

		UpdateSubbandsCoordinates_Scheduler(DSize_Current_X, DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, Last_Level);

	for(int y = (Overlap>>1); y < TDSize_Y; y++)

			WriteSubbands(Data, DSize_Initial_X, DSize_Current_X, TData, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, &y, Last_Level);
}

inline __device__ void WriteSubbands_Scheduler(DATATYPE* Data, int DSize_Current_X,  int DSize_Initial_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int Last_Level, int Incorrect_Vertical_Top, int Incorrect_Vertical_Bottom, int Overlap)
{
	if(			Incorrect_Vertical_Top == 0)		WriteSubbands_Top(Data, DSize_Current_X, DSize_Initial_X, TData, TDSize_Y, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, Last_Level, Overlap);			
	else if(	Incorrect_Vertical_Bottom == 0)		WriteSubbands_Bottom(Data, DSize_Current_X, DSize_Initial_X, TData, TDSize_Y, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, Last_Level, Overlap);	
	else											WriteSubbands_Middle(Data, DSize_Current_X, DSize_Initial_X, TData, TDSize_Y, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, Last_Level, Overlap);

}

inline __device__ void ReadSubbands_iteration(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH, int* index){

	TData[((*index)<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data[*TCoordinate_HL];

	++(*index);

	TData[((*index)<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data[*TCoordinate_LH];
	TData[((*index)<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data[*TCoordinate_LH + (DSize_Current_X>>1)];
}

inline __device__ void ReadSubbands(DATATYPE* Data, int DSize_Current_X, int DSize_Initial_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH)
{		
	for(int y = 0; y < TDSize_Y; y++){
		TData[(y<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data[*TCoordinate_LL];

		ReadSubbands_iteration(Data, DSize_Current_X, TData, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, &y);		
		UpdateSubbandsCoordinates(DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH);
	}		
}

inline __device__ void ReadSubbands_LLaux(DATATYPE* Data, DATATYPE* Data_LL, int DSize_Current_X, int DSize_Initial_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate_LL, int* TCoordinate_HL, int* TCoordinate_LH)
{		
	for(int y = 0; y < TDSize_Y; y++){
		TData[(y<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = Data_LL[*TCoordinate_LL];
		
		ReadSubbands_iteration(Data, DSize_Current_X, TData, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH, &y);		
		UpdateSubbandsCoordinates_LLaux(DSize_Current_X, DSize_Initial_X, TCoordinate_LL, TCoordinate_HL, TCoordinate_LH);
	}		
}

inline __device__ void WriteBlock_int1(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int* index)
{	
	Data[*TCoordinate] = TData[((*index)<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)];
	Data[*TCoordinate+1] = TData[(((*index)<<1)+1)+(SHARED_MEMORY_STRIDE*threadIdx.x)];

	*TCoordinate += DSize_Current_X;
}

__device__ __forceinline__ void st2(DATATYPE2* a, DATATYPE b, DATATYPE c)
{

	#if DWT53_or_DWT97 == 1
		#if DATATYPE_16BITS_or_32BITS == 1
			asm ("st.global.wt.v2.u16 [%0], {%1,%2};" :: "l"(a) , "h"(b), "h"(c));
		#else
			asm ("st.global.wt.v2.u32 [%0], {%1,%2};" :: "l"(a) , "r"(b), "r"(c));
		#endif
	#else
		#if DATATYPE_16BITS_or_32BITS == 1
			asm ("st.global.wt.v2.u16 [%0], {%1,%2};" :: "l"(a) , "h"(b), "h"(c));
		#else
			asm ("st.global.wt.v2.f32 [%0], {%1,%2};" :: "l"(a) , "f"(b), "f"(c));
		#endif
	#endif
}


inline __device__ void WriteBlock_int2(DATATYPE2* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int* index)
{			

	st2(Data+(*TCoordinate), TData[((*index)*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)], TData[((*index)*2) +1+(SHARED_MEMORY_STRIDE*threadIdx.x)]);
	//Data[*TCoordinate] = TData[((*index)*2)+(SHARED_MEMORY_STRIDE*threadIdx.x)];
	*TCoordinate += DSize_Current_X;

}


inline __device__ void WriteBlock(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int* index)
{	
	//WriteBlock_int1(Data, DSize_Current_X, TData, TDSize_Y, TCoordinate, index);
	WriteBlock_int2((DATATYPE2*)Data, DSize_Current_X, TData, TDSize_Y, TCoordinate, index);
}


inline __device__ void WriteBlock_Top(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int Overlap)
{
	for(int y = 0; y < (TDSize_Y - (Overlap>>1)); y++)
		WriteBlock(Data, DSize_Current_X, TData, TDSize_Y, TCoordinate, &y);
}

inline __device__ void WriteBlock_Middle(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int Overlap)
{
	for(int y = 0; y < (Overlap>>1); y++)
		*TCoordinate += DSize_Current_X;
	
	for(int y = (Overlap>>1); y < (TDSize_Y - (Overlap>>1)); y++)
		WriteBlock(Data, DSize_Current_X, TData, TDSize_Y, TCoordinate, &y);
}

inline __device__ void WriteBlock_Bottom(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int Overlap)
{	
	for(int y = 0; y < (Overlap>>1); y++)
		*TCoordinate += DSize_Current_X;

	for(int y = (Overlap>>1); y < TDSize_Y; y++)
		WriteBlock(Data, DSize_Current_X, TData, TDSize_Y, TCoordinate, &y);
}


inline __device__ void WriteBlock_Scheduler(DATATYPE* Data, int DSize_Current_X, REG_DATATYPE* TData, int TDSize_Y, int* TCoordinate, int Incorrect_Vertical_Top, int Incorrect_Vertical_Bottom, int Overlap)
{
	*(TCoordinate)>>=1;

	if(			Incorrect_Vertical_Top == 0)		WriteBlock_Top(Data, DSize_Current_X/2, TData, TDSize_Y, TCoordinate, Overlap);
	else if(	Incorrect_Vertical_Bottom == 0)		WriteBlock_Bottom(Data, DSize_Current_X/2, TData, TDSize_Y, TCoordinate, Overlap);
	else											WriteBlock_Middle(Data, DSize_Current_X/2, TData, TDSize_Y, TCoordinate, Overlap);
	
	*(TCoordinate)<<=1;
}

//END - <DEVICE> DATA MANAGEMENT FUNCTIONS -----------------------------------------------------------------------














//START - <DEVICE> PRE-COMPUTE FUNCTIONS -----------------------------------------------------------------------

//Assign a data block to a warp (compute the coordinates from where the warp will fetch its data)
inline __device__ void InitializeCoordinates(	int DSize_Current_X, int DSize_Initial_X, int DSize_Current_Y, int* TCoordinate_X, int* TCoordinate_Y, int* TCoordinate, int* TCoordinate_LL, 
												int* TCoordinate_HL, int* TCoordinate_LH, int LaneID, int WarpID, int NWarps_X, int NWarps_Y, int WarpWork_Y, int LL_offset, 
												int Special_Level, int Overlap)
{
	int X_effective_work = ((WARPSIZE * NELEMENTS_THREAD_X)-Overlap);
	int Y_effective_work = WarpWork_Y;	
	
	int X_border_coordinate_correction = (((WarpID+1) % NWarps_X)==0)?1:0;
	int Y_border_coordinate_correction = (WarpID> ((NWarps_Y*NWarps_X) - NWarps_X - 1))?1:0;	

	*TCoordinate_X =		(((WarpID % NWarps_X) * X_effective_work) + (LaneID * NELEMENTS_THREAD_X));

	if(X_border_coordinate_correction) 		*TCoordinate_X -= (X_effective_work - (DSize_Current_X % X_effective_work))%X_effective_work + Overlap;

	*TCoordinate_Y =		((WarpID/NWarps_X) * (Y_effective_work));

	if(Y_border_coordinate_correction)		*TCoordinate_Y -= (Y_effective_work - ((DSize_Current_Y- Overlap) % Y_effective_work)) % Y_effective_work ;	

	*TCoordinate =			DSize_Current_X*(*TCoordinate_Y) + *TCoordinate_X;

	if(Special_Level==1) 	*TCoordinate_LL = 		((*TCoordinate_Y>>1)*DSize_Initial_X) + (*TCoordinate_X>>1) ;
	else					*TCoordinate_LL = 		((*TCoordinate_Y>>1)*(DSize_Current_X>>1)) + (*TCoordinate_X>>1) + LL_offset;

	*TCoordinate_HL = 		((*TCoordinate_Y>>1)*DSize_Initial_X) + (*TCoordinate_X>>1) + (DSize_Current_X>>1);
	*TCoordinate_LH = 		(((*TCoordinate_Y>>1) + (DSize_Current_Y>>1))*DSize_Initial_X) + (*TCoordinate_X>>1);

}


//With some image and data block sizes some warps can be assigned to data blocks beyond the image borders. This function check if this happens, and its output will be used in the time to write back the results of the DWT
inline __device__ void IncorrectBorderValues(	int LaneID, int WarpID, int NWarps_X, int NWarps_Y, int* Incorrect_Horizontal, int* Incorrect_Vertical_Top, int* Incorrect_Vertical_Bottom, int Overlap)
{
	if(		(((WarpID % NWarps_X)!=0)		&&	(LaneID <((Overlap>>1)/NELEMENTS_THREAD_X))) ||
			(((WarpID + 1) % NWarps_X)!=0)	&&	(LaneID >(WARPSIZE -1 - ((Overlap>>1)/NELEMENTS_THREAD_X))))
				
				*Incorrect_Horizontal = 1;

	if(		WarpID > (NWarps_X - 1))

				*Incorrect_Vertical_Top = 1;

	if(		WarpID < (NWarps_X*(NWarps_Y-1)))
				
				*Incorrect_Vertical_Bottom = 1;
}

//END - <DEVICE> PRE-COMPUTE FUNCTIONS -----------------------------------------------------------------------














//START - <DEVICE> CUDA KERNELS -----------------------------------------------------------------------

//CUDA KERNEL that computes the forward DWT over an input image. The same kernel is used for both for the 5/3 and 9/7 DWT
__global__ void Kernel_DWT_F(		
									DATATYPE* device_original_image, 
									DATATYPE* device_result_image,
									int DSize_Current_X,
									int DSize_Initial_X,
									int DSize_Current_Y,
									int	NWarps_X, 
									int	NWarps_Y, 
									int WarpWork_Y, 
									int NWarps_Block,
									int Write_LL_offset,
									int Last_Level,
									int write
								)
{
	extern __shared__ int synthetic_shared_memory[];
	
	#if FULLSHARED != 1
		register REG_DATATYPE TData[NELEMENTS_THREAD_Y*NELEMENTS_THREAD_X];		
	#else
		__shared__ REG_DATATYPE TData[SHARED_MEMORY_STRIDE*NTHREADSBLOCK_DWT_F];
	#endif

	#if SHUFFLE != 1
		__shared__ REG_DATATYPE Shared_Data[NTHREADSBLOCK_DWT_F];
	#endif

	int LaneID = 			threadIdx.x & 0x1f;
	int WarpID = 			(((threadIdx.x >> 5) + (blockIdx.x * NWarps_Block)));	
	int Idle_Warp = 		0;
	int TCoordinate_X = 	0;	
	int TCoordinate_Y = 	0;
	int TCoordinate = 		0;
	int TCoordinate_LL =	0;
	int TCoordinate_HL = 	0;
	int TCoordinate_LH = 	0;		

	int Incorrect_Horizontal =	0;
	int Incorrect_Vertical_Top = 0;
	int Incorrect_Vertical_Bottom = 0;

	
	Idle_Warp = 		(WarpID < (NWarps_X*NWarps_Y))?0:1;

	if(Idle_Warp) return;
	
	InitializeCoordinates(	DSize_Current_X, DSize_Initial_X, DSize_Current_Y, &TCoordinate_X, &TCoordinate_Y, &TCoordinate, &TCoordinate_LL, &TCoordinate_HL, &TCoordinate_LH, 
							LaneID, WarpID, NWarps_X, NWarps_Y, WarpWork_Y, Write_LL_offset, Last_Level, OVERLAP);	

	IncorrectBorderValues(	LaneID, WarpID, NWarps_X, NWarps_Y, &Incorrect_Horizontal, &Incorrect_Vertical_Top, &Incorrect_Vertical_Bottom, OVERLAP);

	#if READ == 1				
		ReadBlock2((DATATYPE2*)device_original_image, DSize_Current_X/2, TData, NELEMENTS_THREAD_Y, &TCoordinate);

	#else 
		for(int y = 0; y < NELEMENTS_THREAD_Y; y++){

			TData[(y<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = threadIdx.x + y;
			TData[(y<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = threadIdx.x + y +1;}
	#endif

	#if VERTICAL_COMPUTATION == 1
		#if DWT53_or_DWT97 == 1 	
			Vertical_Filter_Forward_53(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
		#else 
			Vertical_Filter_Forward_97(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
		#endif
	#endif
	#if HORIZONTAL_COMPUTATION == 1
		#if FULLSHARED == 1
			#if DWT53_or_DWT97 == 1 	
					Horizontal_Filter_Forward_53_Full_Shared((VOLATILE REG_DATATYPE*)TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);	
				#else 
					Horizontal_Filter_Forward_97_Full_Shared((VOLATILE REG_DATATYPE*)TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#endif		
		#else
			#if SHUFFLE == 1			

				#if DWT53_or_DWT97 == 1 	
					Horizontal_Filter_Forward_53_Shuffle(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#else 
					Horizontal_Filter_Forward_97_Shuffle(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#endif	
			#else				
				#if DWT53_or_DWT97 == 1 	
					Horizontal_Filter_Forward_53_Shared(TData, Shared_Data, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#else 
					Horizontal_Filter_Forward_97_Shared(TData, Shared_Data, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#endif
			#endif
		#endif
	#endif
	
	if(write)			
		if(Incorrect_Horizontal==0)
			WriteSubbands_Scheduler(device_result_image, DSize_Current_X, DSize_Initial_X, TData, NELEMENTS_THREAD_Y, &TCoordinate_LL, &TCoordinate_HL, &TCoordinate_LH, Last_Level, Incorrect_Vertical_Top, Incorrect_Vertical_Bottom, OVERLAP);			
	
}

//CUDA KERNEL that computes the reverse DWT over an input image. The same kernel is used for both for the 5/3 and 9/7 DWT
__global__ void Kernel_DWT_R(	
									DATATYPE* device_original_image, 
									DATATYPE* device_result_image,
									int DSize_Current_X,
									int DSize_Initial_X,
									int DSize_Current_Y,
									int	NWarps_X, 
									int	NWarps_Y,
									int WarpWork_Y, 
									int NWarps_Block,
									int Read_LL_offset,
									int Write_offset,
									int First_Level,
									int write
								)
{

	extern __shared__ int synthetic_shared_memory[];
	
	#if FULLSHARED != 1
		register REG_DATATYPE TData[NELEMENTS_THREAD_Y*NELEMENTS_THREAD_X];		
	#else
		__shared__ REG_DATATYPE TData[SHARED_MEMORY_STRIDE*NTHREADSBLOCK_DWT_F];
	#endif

	#if SHUFFLE != 1
		__shared__ REG_DATATYPE Shared_Data[NTHREADSBLOCK_DWT_F];
	#endif

	int LaneID = 			threadIdx.x & 0x1f;
	int WarpID = 			(((threadIdx.x >> 5) + (blockIdx.x * NWarps_Block)));
	int TCoordinate_X = 	0;	
	int TCoordinate_Y = 	0;
	int TCoordinate = 		0;
	int TCoordinate_LL = 	0;
	int TCoordinate_HL = 	0;
	int TCoordinate_LH = 	0;	

	int Incorrect_Horizontal =	0;
	int Incorrect_Vertical_Top = 0;
	int Incorrect_Vertical_Bottom = 0;
	int Idle_Warp = 		(WarpID < (NWarps_X*NWarps_Y))?0:1;

	if(Idle_Warp) return;
		
	InitializeCoordinates(	DSize_Current_X, DSize_Initial_X, DSize_Current_Y, &TCoordinate_X, &TCoordinate_Y, &TCoordinate, &TCoordinate_LL, &TCoordinate_HL, &TCoordinate_LH, 
							LaneID, WarpID, NWarps_X, NWarps_Y, WarpWork_Y, Read_LL_offset, First_Level, OVERLAP);	
	
	IncorrectBorderValues(	LaneID, WarpID, NWarps_X, NWarps_Y, &Incorrect_Horizontal, &Incorrect_Vertical_Top, &Incorrect_Vertical_Bottom, OVERLAP);	

	#if READ == 1

		if(First_Level)		ReadSubbands(device_original_image, DSize_Current_X, DSize_Initial_X, TData, NELEMENTS_THREAD_Y, &TCoordinate_LL, &TCoordinate_HL, &TCoordinate_LH);
		else 				ReadSubbands_LLaux(device_original_image, device_result_image-Write_offset, DSize_Current_X, DSize_Initial_X, TData, NELEMENTS_THREAD_Y, &TCoordinate_LL, &TCoordinate_HL, &TCoordinate_LH);

	#else 
		for(int y = 0; y < NELEMENTS_THREAD_Y; y++){

			TData[(y<<1)+(SHARED_MEMORY_STRIDE*threadIdx.x)] = threadIdx.x + y;
			TData[(y<<1)+1+(SHARED_MEMORY_STRIDE*threadIdx.x)] = threadIdx.x + y +1;}
	#endif

	#if HORIZONTAL_COMPUTATION == 1
		#if FULLSHARED == 1
			#if DWT53_or_DWT97 == 1 	
				Horizontal_Filter_Reverse_53_Full_Shared((VOLATILE REG_DATATYPE*)TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
			#else 
				Horizontal_Filter_Reverse_97_Full_Shared((VOLATILE REG_DATATYPE*)TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
			#endif
		#else 
			#if SHUFFLE == 1
				#if DWT53_or_DWT97 == 1 	
					Horizontal_Filter_Reverse_53_Shuffle(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#else 
					Horizontal_Filter_Reverse_97_Shuffle(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#endif
			#else
				#if DWT53_or_DWT97 == 1 	
					Horizontal_Filter_Reverse_53_Shared(TData, Shared_Data, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#else 
					Horizontal_Filter_Reverse_97_Shared(TData, Shared_Data, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
				#endif
			#endif
		#endif
	#endif
	#if VERTICAL_COMPUTATION == 1
		#if DWT53_or_DWT97 == 1 	
			Vertical_Filter_Reverse_53(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
		#else 
			Vertical_Filter_Reverse_97(TData, NELEMENTS_THREAD_Y, NELEMENTS_THREAD_X);
		#endif
	#endif

	if(write)			
		if(Incorrect_Horizontal==0)
			WriteBlock_Scheduler(device_result_image, DSize_Current_X, TData, NELEMENTS_THREAD_Y, &TCoordinate, Incorrect_Vertical_Top, Incorrect_Vertical_Bottom, OVERLAP);	
}

//END - <DEVICE> CUDA KERNELS -----------------------------------------------------------------------














//START - <HOST> DWT FUNCTIONS -----------------------------------------------------------------------

//DWT FORWARD

static inline void DWT_F(int DWT_Levels, int DSize_Initial_X, int DSize_Initial_Y, DATATYPE* DData_Initial, DATATYPE* DData_Final){

	int 		Warps_Row, Warps_Column, warp_vertical_length_work, CUDA_number_blocks, CUDA_number_warps;
	int 		Write_LL_offset = 0;
	int			Last_Level = 0;
	int 		DSize_Current_X = DSize_Initial_X;
	int 		DSize_Current_Y = DSize_Initial_Y;
	DATATYPE*	DData_Initial_aux =	DData_Initial;

//A CUDA kernel is launched for every DWT level
	for(int Current_Level = DWT_Levels; Current_Level>0; --Current_Level)
	{		
		Warps_Row = 					(int)ceil(DSize_Current_X/((float)(WARPSIZE*NELEMENTS_THREAD_X)-OVERLAP));	
		warp_vertical_length_work = 	(NELEMENTS_THREAD_Y)-OVERLAP;
		Warps_Column = 					(int)ceil((((DSize_Current_Y - OVERLAP)/(float)(warp_vertical_length_work))));
		CUDA_number_warps = 			Warps_Row *	Warps_Column;					
		CUDA_number_blocks = 			(int)ceil((CUDA_number_warps*WARPSIZE)/(float)(NTHREADSBLOCK_DWT_F));
		
		Last_Level = (Current_Level == 1) ? 1 : Last_Level;
		Write_LL_offset += (DSize_Current_X*DSize_Current_Y);

		Kernel_DWT_F<<<CUDA_number_blocks,NTHREADSBLOCK_DWT_F, SYNTHETIC_SHARED>>>
									(	
										DData_Initial_aux, 
										DData_Final,									
										DSize_Current_X,
										DSize_Initial_X,
										DSize_Current_Y, 
										Warps_Row, 
										Warps_Column,
										warp_vertical_length_work, 
										NTHREADSBLOCK_DWT_F/WARPSIZE,
										Write_LL_offset, 
										Last_Level,
										WRITE
									);
		
		DData_Initial_aux = DData_Final + (Write_LL_offset);
		
		DSize_Current_X>>=1;
		DSize_Current_Y>>=1;
	}
}

//DWT REVERSE

static inline void DWT_R(int DWT_Levels, int DSize_Initial_X, int DSize_Initial_Y, DATATYPE* DData_Initial, DATATYPE* DData_Final){

	int 		Warps_Row, Warps_Column, warp_vertical_length_work, CUDA_number_blocks, CUDA_number_warps;
	int 		Read_LL_offset = 0;
	int 		Write_offset = 0;
	int			First_Level = 1;
	int 		DSize_Current_X = DSize_Initial_X>>(DWT_Levels-1);
	int 		DSize_Current_Y = DSize_Initial_Y>>(DWT_Levels-1);
	DATATYPE*		DData_Final_aux =	DData_Final;

//A CUDA kernel is launched for every DWT level
	for(int Current_Level = DWT_Levels; Current_Level>0; --Current_Level)
	{
		Warps_Row = 					(int)ceil(DSize_Current_X/((float)(WARPSIZE*NELEMENTS_THREAD_X)-OVERLAP));
		warp_vertical_length_work = 	(NELEMENTS_THREAD_Y)-OVERLAP;
		Warps_Column = 					(int)ceil(((DSize_Current_Y - OVERLAP)/(float)(warp_vertical_length_work)));
		CUDA_number_warps = 			Warps_Row *	Warps_Column;					
		CUDA_number_blocks = 			(int)ceil((CUDA_number_warps*WARPSIZE)/(float)NTHREADSBLOCK_DWT_R);

		Kernel_DWT_R<<<CUDA_number_blocks,NTHREADSBLOCK_DWT_R, SYNTHETIC_SHARED>>>
									(	
										DData_Initial,
										DData_Final_aux,
										DSize_Current_X,
										DSize_Initial_X,
										DSize_Current_Y, 
										Warps_Row, 
										Warps_Column,
										warp_vertical_length_work, 
										NTHREADSBLOCK_DWT_R/WARPSIZE,
										Read_LL_offset,
										Write_offset,
										First_Level,
										WRITE				
									);
		First_Level = 0;		

		Read_LL_offset += (Write_offset - Read_LL_offset);
		Write_offset += (DSize_Current_X*DSize_Current_Y);
		DData_Final_aux = DData_Final + Write_offset;

		DSize_Current_X<<=1;
		DSize_Current_Y<<=1;
	}
}

//END - <HOST> DWT FUNCTIONS -----------------------------------------------------------------------














//START - <HOST> PRE/POST COMPUTE FUNCTIONS -----------------------------------------------------------------------

static inline void Kernel_launcher(int DWT_Direction, int DWT_Levels, int DSize_X, int DSize_Y, DATATYPE* DData_Initial, DATATYPE* DData_Final){
	switch(DWT_Direction){
		case FORWARD:	
			DWT_F(DWT_Levels, DSize_X, DSize_Y, (DATATYPE*) DData_Initial, (DATATYPE*) DData_Final);	
		break;

		case REVERSE:
			DWT_R(DWT_Levels, DSize_X, DSize_Y, DData_Initial, DData_Final);
		break;
	}
}

static inline void Device_memory_allocator(int DWT_Direction, int DWT_Levels, DATATYPE* HData, int HDSize_X, int HDSize_Y, DATATYPE** DData_Initial, DATATYPE** DData_Final, int* DData_Extra){
		
	size_t 		DSize  = HDSize_X*HDSize_Y*sizeof(DATATYPE);
		
	*DData_Extra = 0;	

	for(int i=1; i<DWT_Levels; ++i) *DData_Extra +=  (HDSize_X/(2<<(i-1)))* (HDSize_Y/(2<<(i-1)));	

	hipMalloc ((void**) &(*DData_Initial), DSize);
	hipMalloc ((void**) &(*DData_Final), DSize + ((*DData_Extra)* sizeof(DATATYPE)));

	hipMemcpy(*DData_Initial, (DATATYPE*)HData, DSize, hipMemcpyHostToDevice);
}

static inline void Device_memory_deallocator(int DWT_Direction, DATATYPE* HData, int HDSize_X, int HDSize_Y, DATATYPE* DData_Initial, DATATYPE* DData_Final, int DData_Extra){
	
	size_t 		DSize = HDSize_X*HDSize_Y*sizeof(DATATYPE);
	
	if( (DWT_Direction==REVERSE) )		hipMemcpy(HData, DData_Final + DData_Extra, DSize, hipMemcpyDeviceToHost);
	else 								hipMemcpy(HData, DData_Final, DSize, hipMemcpyDeviceToHost);

	hipFree(DData_Initial);
	hipFree(DData_Final);
	
}


//Apply DWT_Levels over an input data HData of sizes HDSize_X x HDSize_Y. DWT_Direction determines wether it computes the forward or the reverse DWT.
void CUDA_DWT(int DWT_Direction,int DWT_Levels, DATATYPE* HData, int HDSize_X, int HDSize_Y)
{
			
	DATATYPE* 		DData_Initial; 
	DATATYPE* 		DData_Final;
	int				DData_Extra;	
#if FULLSHARED == 1
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
#else
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
#endif
	Device_memory_allocator(DWT_Direction, DWT_Levels, HData, HDSize_X, HDSize_Y, &DData_Initial, &DData_Final, &DData_Extra);
	Kernel_launcher(DWT_Direction, DWT_Levels, HDSize_X, HDSize_Y, DData_Initial, DData_Final);
	Device_memory_deallocator(DWT_Direction, HData, HDSize_X, HDSize_Y, DData_Initial, DData_Final, DData_Extra);
}	

//END - <HOST> PRE/POST COMPUTE FUNCTIONS -----------------------------------------------------------------------













	
