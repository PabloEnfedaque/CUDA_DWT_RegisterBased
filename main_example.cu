#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include "RBased_DWT_common.h"


#if DATATYPE_16BITS_or_32BITS == 1
	#if DWT53_or_DWT97 == 0
		#define TESTDATATYPE float
	#else 		
		#define TESTDATATYPE DATATYPE
	#endif
#else 
		#define TESTDATATYPE DATATYPE
#endif


void CUDA_DWT(int, int, DATATYPE* , int , int);


void Check(TESTDATATYPE* image1, DATATYPE* image2, int xSize, int ySize, long int *correct_flag, long int *wrong_count){

	for(int i=0;(i<ySize);i++){	
		for(int j=0;(j<xSize);j++) {


		#if DWT53_or_DWT97 == 1
			if(((DATATYPE)image1[(xSize*i)+j]) != ((int)image2[(xSize*i)+j])){
				
		#else 

			#if DATATYPE_16BITS_or_32BITS == 1

			if(abs(((DATATYPE)(image1[(xSize*i)+j])) - (image2[(xSize*i)+j]))>16){			

			#else 
				
			if(abs(((DATATYPE)(image1[(xSize*i)+j])) - (image2[(xSize*i)+j]))>1){			
	
			#endif

		#endif

				if(*correct_flag==-1){
					*correct_flag = (xSize*i)+j;
					*wrong_count= *wrong_count +1 ;
				}
				else ++(*wrong_count);
			}
		}
	}
	
}


void Print_Results(int xSize,int ySize,long int correct_flag, long int wrong_count, int levels){

	if(correct_flag==-1)	printf("TEST	image size: %dx%d	with %d levels	>>>	OK\n", xSize, ySize, levels);
	else printf("TEST	image size: %dx%d	with %d levels	>>>	ERROR ...... first error position:  row %ld, column %ld //  number mismatches: %ld\n", 
					xSize, ySize, levels, correct_flag/xSize, correct_flag%xSize, wrong_count);
}

int main(int argc, char** argv)
{
	int aux;
	int ySize, xSize;
	int levels = LEVELS;

	printf("\nComputing the dwt...\n\n");		

		printf("------------------------------------------------------------------------------------\n");	
		#if DWT53_or_DWT97 == 1
			printf("				DWT 53\n");	
		#else 
			printf("				DWT 97\n");
		#endif
		printf("------------------------------------------------------------------------------------\n");
		printf("*** Comparing forward versus reverse DWT results, with some random generated samples ***\n");
		printf("------------------------------------------------------------------------------------\n\n");

	for(int i= 0;i<NEXPERIMNETS;i++)
	{	

		aux= EXPERIMENT_INI;
		ySize = ((i)*EXPERIMENT_INCREMENT)+EXPERIMENT_INI;
		xSize = ((i)*EXPERIMENT_INCREMENT)+EXPERIMENT_INI;
		
		srand(time(NULL));


		TESTDATATYPE *image1 = (TESTDATATYPE*)malloc((ySize) * (xSize) * sizeof(TESTDATATYPE));
		DATATYPE *image2 = (DATATYPE*)malloc((ySize) * (xSize) * sizeof(DATATYPE));


		for(int i=0;i<xSize;i++){	for(int j=0;j<ySize;j++){ 
			
			aux = (rand() % 255) -128; 

			image1[(xSize*i)+j]= aux; 
			image2[(xSize*i)+j]= aux; 

		}}		

		long int correct_flag=-1;
		long int wrong_count=0;

		CUDA_DWT(FORWARD, levels, image2 , xSize , ySize);
		CUDA_DWT(REVERSE, levels, image2 , xSize , ySize);		

		Check(image1, image2, xSize, ySize, &correct_flag, &wrong_count);
		Print_Results(xSize, ySize, correct_flag, wrong_count, levels);

				
		free(image1);
		free(image2);				
		
	}

	printf("\n");

	
	hipDeviceReset(); 
	return(0);
}
